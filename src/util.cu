#include "hip/hip_runtime.h"
#include <cstdlib>
#include "ant_colony.h"

int randint(int n)
{
    // return number from {0,1,... n-1}
    return rand() % n;
}

int randprob(double p)
{
    // return 1 with probability p
    double d = ((double)rand()) / RAND_MAX;
    return d < p;
}

double rand01()
{
    return ((double)randint(100)) / 100.0;
}

void coords_to_edge(string infile, string outfile){
    // accepts format
    // n
    // <node> <x coord> <y coord>
    // outputs our format (from graph read)
    ifstream in(infile.c_str(), ios::in);
    ofstream out(outfile.c_str());
    int n;
    in>>n;
    vd x(n),y(n);
    int r;
    for(int i=0;i<n;i++) in>>r>>x[i]>>y[i];
    out<<n<<" "<<(n*(n-1))/2<<"\n";
    for(int i=0;i<n;i++) for(int j=i+1;j<n;j++) {
        double d = sqrt((x[i]-x[j])*(x[i]-x[j]) + (y[i]-y[j])*(y[i]-y[j]));
        out<<i<<" "<<j<<" "<<d<<"\n";
    }
}

void adjmat_to_edge(string infile, string outfile){
    // accepts format
    // n
    // a00 a01 ....
    // .
    // outputs our format (from graph read)
    ifstream in(infile.c_str());
    ofstream out(outfile.c_str());
    int n;
    in>>n;
    out<<n<<" "<<n*n<<"\n";
    for(int i=0;i<n;i++) for(int j=0;j<n;j++){
        double w;
        in>>w;
        out<<i<<" "<<j<<" "<<w<<"\n";
    }
}

Graph input_graph(){
    Graph g;
    int n,m;
    cin >> n >> m;
    g.n=n;
    g.adj = adj_t(n);
    for (int i = 0; i < m; i++)
    {
        int u, v;
        double w;
        cin >> u >> v >> w;
        g.add_edge(u, v, w);
    }
    return g;
}
