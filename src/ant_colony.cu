#include "hip/hip_runtime.h"
#include <iostream>
#include "ant_colony.h"
#include <hiprand/hiprand_kernel.h>

double PH_EVAP = 0.5;
double ALPHA = 1;
double BETA = 1;

__device__ void gen_sol(int* path,double* cost,int n,double* gadj, double* pmap, hiprandState &s){
    int *reached = (int*) malloc(n*sizeof(int));
    for(int i=0;i<n;i++) reached[i]=0;
    int *vert= (int*) malloc(n*sizeof(int));
    double *cprob= (double*) malloc(n*sizeof(double));
    reached[0] = 1;
    path[0] = 0;
    *cost = 0;
    for (int i = 1; i < n; i++)
    {
        int u=path[i-1];
        // pick next vertex - i already filled
        // weight by pmap[i][j] / g.adj[i][j].second
        // TODO weight by pmap[i][j]^alpha / g.adj[i][j].second^beta
        int sz=0;
        double tot=0.0;
        for(int v=0; v<n;v++){
            if(reached[v]) continue;       
            //printf("u %d v %d p %f g %f\n",u,v,pmap[u*n+v],gadj[u*n+v]);
            double p = pmap[u*n+v] / gadj[u*n + v];
            
            cprob[sz] = p+tot;
            vert[sz]=v;
            sz++;
            tot+=p;
        
        }
        double rnd =hiprand_uniform(&s)*tot;
        //printf("rnd %f\n",rnd);
        for (int k = 0; k < sz; k++)
        {
            if (rnd < cprob[k])
            {
                int v=vert[k];
                //printf("picking vertex %d\n",v);
                path[i]=v;
                reached[v]=1;
                *cost += gadj[u*n+v];
                break;
            }
        }
        //path[i]=u+1;
        //reached[i]=1;
        //continue;
    }   
    // add path[n-1]-path[0] cost
    *cost += gadj[path[n-1]*n + path[0]];
}


void ph_update(int n,int* path, double* cost, double* pmap){
    int v = path[0],u;
    for(int i=1; i<n;i++){
        u = path[i];
        // update ph at edge v-u
        pmap[v*n+u] = (1- PH_EVAP)*pmap[v*n+u] + 1/(*cost);
        v = u;
    }
    u = 0;
    pmap[v*n+u] = (1- PH_EVAP)*pmap[v*n+u] + 1/(*cost);
}



__global__ void test_k(int n, int *paths, double* costs, double *gadj, double *p, int seed){
    int i = threadIdx.x;
    //printf("id: %d %d\n",i, n);
    if(i >=n ) return;
    
    hiprandState s;
    hiprand_init(i, seed, 0, &s);

    int *rpath = paths+i*n;
    double *rcost = costs+i;
    for(int i=0;i<n;i++) rpath[i]=-1;
    gen_sol(rpath, rcost, n,gadj, p, s);
    //printf("id %d cost %f cost' %f\n",i,*rcost,costs[i]);

}



int ant_colony_opt_tsp(int n, double* gadj, int *path, double &cost)
{
    /*
        Ant Colony System, specific to TSP
        Takes weighted graph input <g>, fills up best found path in <path>
    */
    int NI = 100,
        NA = 100;
        
    double *p = new double[n*n];
    for(int i=0;i<n*n;i++) p[i]=1e-5;
    cost=1.0e10;
    for (int iter = 0; iter < NI; iter++)
    {
        int *paths = new int[NA*n]; // paths[ant] = ant's path
        double *costs = new double[NA];
        int *paths_d;
        double *costs_d, *gadj_d, *p_d;
        hipMalloc(&paths_d, NA*n*sizeof(int));
        hipMalloc(&costs_d, NA*sizeof(double));
        hipMalloc(&gadj_d, n*n*sizeof(double));
        hipMalloc(&p_d, n*n*sizeof(double));

        hipMemcpy(gadj_d, gadj, n*n*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(p_d, p, n*n*sizeof(double), hipMemcpyHostToDevice);

        test_k<<<1,NA>>>(n,paths_d, costs_d, gadj_d, p_d,iter);

        hipDeviceSynchronize();

        hipMemcpy(paths, paths_d, NA*n*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(costs, costs_d, NA*sizeof(double), hipMemcpyDeviceToHost);
        
        for(int i=0;i<0;i++){
            print_vi(paths+i*n,n);
            printf(", cost=%f\n",costs[i]);
        }

        int mini=-1;
        for (int i= 0; i< NA; i++)
        {
            int *rpath = paths+i*n;
            double *rcost = costs+i;
            if(*rcost < cost){
                mini=i;
                cost = *rcost;
            }   
            ph_update(n,rpath, rcost, p);
        }
        if(mini!=-1){
            for(int i=0;i<n;i++) path[i]=paths[mini*n+i];
        }

    }
    return 0;
}
