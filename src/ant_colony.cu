#include "hip/hip_runtime.h"
#include <iostream>
#include "ant_colony.h"
#include <ctime>
#include <hiprand/hiprand_kernel.h>

double ALPHA = 1;
double BETA = 1;

__device__ void gen_sol(int* path,double* cost,int n,double* gadj, double* pmap, hiprandState &s){
    int *reached = (int*) malloc(n*sizeof(int)); // reached
    for(int i=0;i<n;i++) reached[i]=0;
    int *vert= (int*) malloc(n*sizeof(int));
    double *cprob= (double*) malloc(n*sizeof(double));
    reached[0] = 1;
    path[0] = 0;
    *cost = 0;
    for (int i = 1; i < n; i++)
    {
        int u=path[i-1];
        // pick next vertex - i already filled
        // weight by pmap[i][j] / g.adj[i][j].second
        // TODO weight by pmap[i][j]^alpha / g.adj[i][j].second^beta
        int sz=0;
        double tot=0.0;
        for(int v=0; v<n;v++){
            if(reached[v]) continue;       
            //printf("u %d v %d p %f g %f\n",u,v,pmap[u*n+v],gadj[u*n+v]);
            double p = pmap[u*n+v] / gadj[u*n + v];
            
            cprob[sz] = p+tot;
            vert[sz]=v;
            sz++;
            tot+=p;
        
        }
        double rnd =hiprand_uniform(&s)*tot;
        //printf("rnd %f\n",rnd);
        for (int k = 0; k < sz; k++)
        {
            if (rnd < cprob[k])
            {
                int v=vert[k];
                //printf("picking vertex %d\n",v);
                path[i]=v;
                reached[v]=1;
                *cost += gadj[u*n+v];
                break;
            }
        }
        //path[i]=u+1;
        //reached[i]=1;
        //continue;
    }   
    // add path[n-1]-path[0] cost
    *cost += gadj[path[n-1]*n + path[0]];
}

__device__ void atomicUpdate (double* pval_addr, double cost)  {  
	unsigned long long int* address_as_ull = (unsigned long long int*) pval_addr; 
  	unsigned long long int old = *address_as_ull, assumed, newVal; 
	double rCost = (1.0 / cost);
	double PH_EVAP = 0.5;
	double PH_STAY = 1 - PH_EVAP;

  	do { 
 		assumed = old;
		newVal = __double_as_longlong(rCost + PH_STAY * __longlong_as_double(assumed));
 		old = atomicCAS(address_as_ull, assumed, newVal); 
  	} while (assumed != old);
 
	//return __longlong_as_double(old);
}      


/*void ph_update(int n,int* path, double* cost, double* pmap){
    //printf("updating: ");
    //print_vi(path, n);
    //printf(", cost: %f\n", *cost);
    
    int v = path[0],u;
    for(int i=1; i<n;i++){
        u = path[i];
        // update ph at edge v-u
        //printf("update edge %d-%d\n",v,u);
        pmap[v*n+u] = (1- PH_EVAP)*pmap[v*n+u] + 1/(*cost);
        v = u;
    }
    u = 0;
    //printf("update edge %d-%d\n",v,u);
    pmap[v*n+u] = (1- PH_EVAP)*pmap[v*n+u] + 1/(*cost);
}*/


__device__ void ph_update (int n,int* path, double cost, double* pmap){
    //printf("updating: ");
    //print_vi(path, n);
    //printf(", cost: %f\n", cost);
    
    int i;
    for (i = 0; i < n-1; i++) {
        atomicUpdate(&pmap[path[i] * n + path[i+1]], cost);
    }
    atomicUpdate(&pmap[path[n-1] * n], cost);
}



__global__ void test_k(int n, int *paths, double* costs, double *gadj, double *p, int seed, double *cost, int *mini, int *path){
    int i = threadIdx.x;
    //printf("id: %d %d\n",i, n);
    if(i >=n ) return;
    
    hiprandState s;
    hiprand_init(i, seed, 0, &s);

    int *rpath = paths+i*n;
    double *rcost = costs+i;
    for(int i=0;i<n;i++) rpath[i]=-1;
    gen_sol(rpath, rcost, n,gadj, p, s);
    //printf("id %d cost %f cost' %f\n",i,*rcost,costs[i]);

    
    unsigned long long oldCost, assumedCost, newCost;
    int oldMini, assumedMini, newMini;
    
    if(*rcost < *cost){
	unsigned long long *ullCost = (unsigned long long *) cost;
	oldCost = *ullCost;
	do { 
 	  assumedCost = oldCost;
	  newCost = __double_as_longlong(*rcost);
 	  oldCost = atomicCAS(ullCost, assumedCost, newCost); 
  	} while (assumedCost != oldCost);

	oldMini = *mini;
	do { 
 	  assumedMini = oldMini;
	  newMini = i;
 	  oldMini = atomicCAS(mini, assumedMini, newMini); 
  	} while (assumedMini != oldMini);
    }
    __syncthreads();

    if (*mini != -1) {
       if(i < n) {
            path[i]=paths[(*mini)*n+i];
       }
    }
    __syncthreads();

    ph_update(n, rpath, *rcost, p);
}



int ant_colony_opt_tsp(int n, double* gadj, int *path, double &cost)
{
    /*
        Ant Colony System, specific to TSP
        Takes weighted graph input <g>, fills up best found path in <path>
    */
    int NI = 1000,
        NA = 10;

    clock_t st = clock();
        
    double *p = new double[n*n];
    for(int i=0;i<n*n;i++) p[i]=1e-5;
    cost=1.0e10;

    int *paths = new int[NA*n]; // paths[ant] = ant's path
    double *costs = new double[NA];
    int *paths_d;
    double *costs_d, *gadj_d, *p_d;
    double *cost_d;
    int *mini_d;
    int *path_d;

    hipMalloc(&paths_d, NA*n*sizeof(int));
    hipMalloc(&costs_d, NA*sizeof(double));
    hipMalloc(&gadj_d, n*n*sizeof(double));
    hipMalloc(&p_d, n*n*sizeof(double));
    hipMalloc(&cost_d, sizeof(double));
    hipMalloc(&mini_d, sizeof(double));
    hipMalloc(&path_d, n*sizeof(int));

    hipMemcpy(gadj_d, gadj, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(p_d, p, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cost_d, &cost, sizeof(double), hipMemcpyHostToDevice);

    for (int iter = 0; iter < NI; iter++)
    {
        /*int *paths = new int[NA*n]; // paths[ant] = ant's path
        double *costs = new double[NA];
        int *paths_d;
        double *costs_d, *gadj_d, *p_d;
	double *cost_d;

        hipMalloc(&paths_d, NA*n*sizeof(int));
        hipMalloc(&costs_d, NA*sizeof(double));
        hipMalloc(&gadj_d, n*n*sizeof(double));
        hipMalloc(&p_d, n*n*sizeof(double));
	hipMalloc(&cost_d, sizeof(double));

        hipMemcpy(gadj_d, gadj, n*n*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(p_d, p, n*n*sizeof(double), hipMemcpyHostToDevice);*/

	int mini = -1;
	hipMemcpy(mini_d, &mini, sizeof(int), hipMemcpyHostToDevice);

        test_k<<<1,NA>>>(n,paths_d, costs_d, gadj_d, p_d,iter, cost_d, mini_d, path_d);

        hipDeviceSynchronize();

        //hipMemcpy(paths, paths_d, NA*n*sizeof(int), hipMemcpyDeviceToHost);
        //hipMemcpy(costs, costs_d, NA*sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(&cost, cost_d, sizeof(double), hipMemcpyDeviceToHost);

        
        /*for(int i=0;i<0;i++){
            print_vi(paths+i*n,n);
            printf(", cost=%f\n",costs[i]);
        }*/

        /*for (int i= 0; i< NA; i++)
        {
            double *rcost = costs+i;
            if(*rcost < cost){
                mini=i;
            }   
        */

	/*hipMemcpy(&mini, mini_d, sizeof(int), hipMemcpyDeviceToHost);
        if(mini!=-1){
            for(int i=0;i<n;i++) path[i]=paths[mini*n+i];
        }*/


        /*delete[] paths;
        delete[] costs;
        hipFree(gadj_d);
        hipFree(p_d);
        hipFree(paths_d);
        hipFree(costs_d);
	hipFree(cost_d);*/
    }

    hipMemcpy(path, path_d, n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&cost, cost_d, sizeof(double), hipMemcpyDeviceToHost);

    delete[] paths;
    delete[] costs;
    hipFree(gadj_d);
    hipFree(p_d);
    hipFree(paths_d);
    hipFree(costs_d); 
    hipFree(cost_d);
    hipFree(path_d);

    delete[] p;
    clock_t en = clock();
    double t = (double)(en-st)/CLOCKS_PER_SEC;
    printf("Evaluated %d ants each\n",NI*NA);
    printf("Took time %f\n",t);
    return 0;
}
