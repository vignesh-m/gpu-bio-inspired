#include <cstdio>
#include <iostream>
#include "ant_colony.h"

int main(){
    srand(time(NULL));

    cout << "Hello\n";
    Graph g = input_graph();
    print_graph(g);

    vi path;
    double cost;
    ant_colony_opt_tsp(path, cost, g);

    print_vi(path);
    cout<<" "<<cost<<"\n";

    printf("pls\n");
    hipDeviceSynchronize();
}
