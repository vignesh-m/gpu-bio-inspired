#include <cstdio>
#include <iostream>
#include "ant_colony.h"

int main(){
    srand(time(NULL));

    cout << "Hello\n";
    Graph g = input_graph();
    int n=g.n;
    //print_graph(g);

    int *path = new int[n];
    double cost;
    double* gadj = new double[n*n];
    to_1d(gadj,g);
    ant_colony_opt_tsp(g.n, gadj, path, cost);

    print_vi(path,n);
    cout<<" "<<cost<<"\n";

    hipDeviceSynchronize();
}
